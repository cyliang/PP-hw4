/**********************************************************************
* DESCRIPTION:
*   Serial Concurrent Wave Equation - C Version
*   This program implements the concurrent wave equation
*********************************************************************/
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

#define cudaFn(fn,...) { \
	hipError_t __cudaStatus = fn(__VA_ARGS__); \
	if (__cudaStatus != hipSuccess) { \
		fprintf(stderr, "Failed when calling cuda function \"%s\"!\n%s\n", #fn, hipGetErrorString(__cudaStatus)); \
		exit(1); \
	} \
}
#define kernelFn(fn,block_count,thread_count,...) { \
	fn<<<block_count, thread_count>>>(__VA_ARGS__); \
	hipError_t __cudaStatus = hipGetLastError(); \
	if (__cudaStatus != hipSuccess) { \
		fprintf(stderr, "%s launch failed: %s\n", #fn, hipGetErrorString(__cudaStatus)); \
		exit(1); \
	} \
}

float result[MAXPOINTS + 2];

/**********************************************************************
*	Checks input values from parameters
*********************************************************************/
__host__ void check_param(int *tpoints, int *nsteps) {
	char tchar[20];

	/* check number of points, number of iterations */
	while ((*tpoints < MINPOINTS) || (*tpoints > MAXPOINTS)) {
		printf("Enter number of points along vibrating string [%d-%d]: "
			   , MINPOINTS, MAXPOINTS);
		scanf("%s", tchar);
		*tpoints = atoi(tchar);
		if ((*tpoints < MINPOINTS) || (*tpoints > MAXPOINTS))
			printf("Invalid. Please enter value between %d and %d\n",
			MINPOINTS, MAXPOINTS);
	}
	while ((*nsteps < 1) || (*nsteps > MAXSTEPS)) {
		printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
		scanf("%s", tchar);
		*nsteps = atoi(tchar);
		if ((*nsteps < 1) || (*nsteps > MAXSTEPS))
			printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
	}

	printf("Using points = %d, steps = %d\n", *tpoints, *nsteps);

}

/**********************************************************************
*      Calculate new values using wave equation
*********************************************************************/
__device__ float do_math(float values, float oldval) {
	float dtime, c, dx, tau, sqtau;

	dtime = 0.3;
	c = 1.0;
	dx = 1.0;
	tau = (c * dtime / dx);
	sqtau = tau * tau;
	return (2.0 * values) - oldval + (sqtau *  (-2.0)*values);
}

__global__ void update(float *values, int tpoints, int nsteps) {
	/**********************************************************************
	*     Initialize points on line
	*********************************************************************/
	int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
	float x, fac;
	float nowval,	/* values at time t */ 
		oldval,		/* values at time (t-dt) */
		newval;		/* values at time (t+dt) */

	/* Calculate initial values based on sine curve */
	fac = 2.0 * PI;
	x = (-1.0 + idx) / (float) (tpoints - 1);

	nowval = sin(fac * x);

	/* Initialize old values array */
	oldval = nowval;




	/**********************************************************************
	*     Update all values along line a specified number of times
	*********************************************************************/
	int i;

	/* Update values for each time step */
#pragma unroll 1000
	for (i = 0; i < nsteps; i++) {
		/* Update points along line for this time step */
		newval = do_math(nowval, oldval);

		/* Update old values with new values */
		oldval = nowval;
		nowval = newval;
	}

	/* global endpoints */
	if ((idx == 1) || (idx == tpoints))
		nowval = 0.0;

	values[idx] = nowval;
}

/**********************************************************************
*     Print final results
*********************************************************************/
__host__ void printfinal(float values[], int tpoints) {
	int i;

	for (i = 1; i <= tpoints; i++) {
		printf("%6.4f ", values[i]);
		if (i % 10 == 0)
			printf("\n");
	}
}

/**********************************************************************
*	Main program
*********************************************************************/
__host__ int main(int argc, char *argv[]) {
	int nsteps,                 	/* number of time steps */
		tpoints; 	     		/* total points along string */
	float *values;

	if (argc != 3) {
		printf("Usage: %s <number_of_points> <number_of_time_steps>\n", argv[0]);
		return 1;
	}
	sscanf(argv[1], "%d", &tpoints);
	sscanf(argv[2], "%d", &nsteps);
	check_param(&tpoints, &nsteps);

	int blockCount = tpoints / 1024 + 1;
	int arySize = (blockCount * 1024 + 1) * sizeof(float);
	cudaFn(hipMalloc, &values, arySize);

	printf("Initializing points on the line...\n");
	printf("Updating all points for all time steps...\n");
	kernelFn(update, blockCount, 1024, values, tpoints, nsteps);

	printf("Printing final results...\n");
	cudaFn(hipMemcpy, result, values, arySize, hipMemcpyDeviceToHost);
	printfinal(result, tpoints);
	printf("\nDone.\n\n");

	cudaFn(hipFree, values);
	cudaFn(hipDeviceReset);
	return 0;
}
